#include <stdio.h>
#include <hip/hip_runtime.h>
#include "include/reduce_kernels.h"

int main() {
    const int N = 1000;
    float* h_input = new float[N];
    for (int i = 0; i < N; ++i) h_input[i] = i;

    float h_output = 0.0f;
    float *d_input, *d_output;
    hipMalloc(&d_input, sizeof(float) * N);
    hipMalloc(&d_output, sizeof(float));
    hipMemcpy(d_input, h_input, sizeof(float) * N, hipMemcpyHostToDevice);

    printf("== Warp-Level Reduce ==\n");
    launch_warp_reduce(d_input, d_output, 32);
    hipMemcpy(&h_output, d_output, sizeof(float), hipMemcpyDeviceToHost);
    printf("Result: %.1f\n", h_output);  // 0+1+...+31 = 496

    // 只有第一个块
    printf("== Block-Level Reduce ==\n");
    launch_block_reduce(d_input, d_output, N);
    hipMemcpy(&h_output, d_output, sizeof(float), hipMemcpyDeviceToHost);
    printf("Result: %.1f\n", h_output);

    printf("== Grid-Level Reduce (Sum) ==\n");
    launch_grid_reduce_op(d_input, d_output, N, SumOp());
    hipMemcpy(&h_output, d_output, sizeof(float), hipMemcpyDeviceToHost);
    printf("Sum Result: %.1f\n", h_output);

    printf("== Grid-Level Reduce (Max) ==\n");
    launch_grid_reduce_op(d_input, d_output, N, MaxOp());
    hipMemcpy(&h_output, d_output, sizeof(float), hipMemcpyDeviceToHost);
    printf("Max Result: %.1f\n", h_output);

    printf("== Grid-Level Reduce (Product) ==\n");
    launch_grid_reduce_op(d_input, d_output, N, ProdOp());
    hipMemcpy(&h_output, d_output, sizeof(float), hipMemcpyDeviceToHost);
    printf("Product Result: %.1f\n", h_output);

    hipFree(d_input); hipFree(d_output); delete[] h_input;
    return 0;
}
